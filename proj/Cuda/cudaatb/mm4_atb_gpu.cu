
#include <hip/hip_runtime.h>
__global__ void aTb_gpu(const float *__restrict__ A, const float *__restrict__ B, float *__restrict__ C, int Ni, int Nj, int Nk)
{
  int i, j, k;

  for (i = 0; i < Ni; i++) 
   for (j = 0; j < Nj; j++)
    C[i*Nj+j]=0.0;
  for (i = 0; i < Ni; i++)
   for (j = 0; j < Nj; j++)
    for (k = 0; k < Nk; k++)
// C[i][j] = C[i][j] + A[k][i]*B[k][j];
     C[i*Nj+j]=C[i*Nj+j]+A[k*Ni+i]*B[k*Nj+j];
}