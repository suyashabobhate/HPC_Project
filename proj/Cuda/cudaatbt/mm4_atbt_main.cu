
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <math.h>
#include <stdlib.h>
#include <sys/time.h>
#define threshold 0.0001
#define BLOCK_SIZE 16
#define FIXME 1

void checkCUDAError(const char *msg);

hipEvent_t start, stop;
float tstart, elapsedTime;

__global__ void atbt_gpu(const float *__restrict__ A, const float *__restrict__ B, float *__restrict__ C, int Ni, int Nj, int Nk);
__global__ void atbt_gpu_kunroll(const float *__restrict__ A, const float *__restrict__ B, float *__restrict__ C, int Ni, int Nj, int Nk);
__global__ void atbt_gpu_junroll(const float *__restrict__ A, const float *__restrict__ B, float *__restrict__ C, int Ni, int Nj, int Nk);
__global__ void atbt_gpu_iunroll(const float *__restrict__ A, const float *__restrict__ B, float *__restrict__ C, int Ni, int Nj, int Nk);
__global__ void atbt_gpu_junroll8(const float *__restrict__ A, const float *__restrict__ B, float *__restrict__ C, int Ni, int Nj, int Nk);
__global__ void atbt_gpu_ijunroll(const float *__restrict__ A, const float *__restrict__ B, float *__restrict__ C, int Ni, int Nj, int Nk);

void aTbT_seq(const float *__restrict__ A, const float *__restrict__ B, float *__restrict__ C, int Ni, int Nj, int Nk)
{
  int i, j, k;

  for (i = 0; i < Ni; i++)
   for (j = 0; j < Nj; j++)
    for (k = 0; k < Nk; k++)
// C[i][j] = C[i][j] + A[k][i]*B[j][k];
     C[i*Nj+j]=C[i*Nj+j]+A[k*Ni+i]*B[j*Nk+k];
}

int main(int argc, char *argv[]){

  float *h_A, *h_B, *h_C, *h_Cref, *d_A, *d_B, *d_C;
  int i,j,k;
  int Ni, Nj, Nk;

  if(argc >= 3) {
        Ni = atoi(argv[1]);
        Nj = atoi(argv[2]);
        Nk = atoi(argv[3]);
  }

  h_A = (float *) malloc(sizeof(float)*Ni*Nk);
  h_B = (float *) malloc(sizeof(float)*Nk*Nj);
  h_C = (float *) malloc(sizeof(float)*Ni*Nj);
  h_Cref = (float *) malloc(sizeof(float)*Ni*Nj);;

  for (i=0; i<Ni; i++)
   for (k=0; k<Nk; k++)
    h_A[k*Ni+i] = rand();
  for (k=0; k<Nk; k++)
   for (j=0; j<Nj; j++)
    h_B[k*Nj+j] = rand();

  
 // Allocate device memory and copy input data over to GPU
  hipMalloc(&d_A, Ni*Nk*sizeof(float));
  hipMalloc(&d_B, Nk*Nj*sizeof(float));
  hipMalloc(&d_C, Ni*Nj*sizeof(float));
  checkCUDAError("hipMalloc failure");
  hipMemcpy(d_A, h_A, Ni*Nk*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, Nk*Nj*sizeof(float), hipMemcpyHostToDevice);
  checkCUDAError("hipMemcpy H2D transfer failure");

  dim3 block(BLOCK_SIZE,BLOCK_SIZE);  
  dim3 grid(ceil(Ni/(float)BLOCK_SIZE),ceil(Nj/(float)BLOCK_SIZE));
  dim3 grid2(ceil(Ni/(float)BLOCK_SIZE),ceil(Nj/4/(float)BLOCK_SIZE)); // j unroll 4
  dim3 grid3(ceil(Ni/4/(float)BLOCK_SIZE),ceil(Nj/(float)BLOCK_SIZE)); // i unroll 4
  dim3 grid4(ceil(Ni/(float)BLOCK_SIZE),ceil(Nj/8/(float)BLOCK_SIZE)); // j unroll 8
  dim3 grid5(ceil(Ni/4/(float)BLOCK_SIZE),ceil(Nj/4/(float)BLOCK_SIZE)); // ij unroll 4
  for(int version=0; version<6; version++)
  {
   for(i=0;i<Ni;i++) for(j=0;j<Nj;j++) h_Cref[i*Nj+j] = 0;
   aTbT_seq(h_A,h_B,h_Cref,Ni,Nj,Nk);
    for(int trial=0;trial<1;trial++)
    {
     for(i=0;i<Ni;i++) for(j=0;j<Nj;j++) h_C[i*Nj+j] = 0; 
      printf("Trial %d: ",trial);
      hipEventCreate(&start);
      hipEventCreate(&stop);
      hipEventRecord(start);
      // Launch kernel
      switch (version) {
      case 0: atbt_gpu<<<grid, block>>>(d_A, d_B, d_C,Ni,Nj,Nk); printf("ATBT "); break;
      case 1: atbt_gpu_kunroll<<<grid, block>>>(d_A, d_B, d_C,Ni,Nj,Nk); printf("ATBT K Unroll ");break;
      case 2: atbt_gpu_junroll<<<grid2, block>>>(d_A, d_B, d_C,Ni,Nj,Nk); printf("ATBT J Unroll ");break;
      case 3: atbt_gpu_iunroll<<<grid3, block>>>(d_A, d_B, d_C,Ni,Nj,Nk); printf("ATBT I Unroll "); break;
      case 4: atbt_gpu_junroll8<<<grid4, block>>>(d_A, d_B, d_C,Ni,Nj,Nk); printf("ATBT J Unroll by 8 "); break;
      case 5: atbt_gpu_ijunroll<<<grid5, block>>>(d_A, d_B, d_C,Ni,Nj,Nk); printf("ATBT IJ Unroll by 4 "); break;
      }
      checkCUDAError("GPU kernel launch failure");
      hipEventRecord(stop);
      hipEventSynchronize(stop);
      hipEventElapsedTime(&elapsedTime, start,stop);
      hipDeviceSynchronize();
      // Copy results back to host
      hipMemcpy(h_C, d_C, Ni*Nj*sizeof(float), hipMemcpyDeviceToHost);
      checkCUDAError("hipMemcpy D2H");
      for (int i = 0; i < Ni*Nj; i++) if (fabs((h_C[i]-h_Cref[i])/h_Cref[i])>threshold) {printf("Error: mismatch at linearized index %d, was: %f, should be: %f\n", i, h_C[i], h_Cref[i]); return -1;}
      printf("GFLOPS: %.2f\n",2.0e-6*Ni*Nj*Nk/elapsedTime);
     }
  }
  return 0;
}

void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err)
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }
}

