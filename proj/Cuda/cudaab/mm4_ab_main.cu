
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <math.h>
#include <stdlib.h>
#include <sys/time.h>
#define threshold 0.0001
#define BLOCK_SIZE 16
#define FIXME 1

void checkCUDAError(const char *msg);

hipEvent_t start, stop;
float tstart, elapsedTime;

__global__ void ab_gpu(const float *__restrict__ A, const float *__restrict__ B, float *__restrict__ C, int Ni, int Nj, int Nk);


void ab_seq(const float *__restrict__ A, const float *__restrict__ B, float *__restrict__ C, int Ni, int Nj, int Nk)
{
  int i, j, k;

  for (i = 0; i < Ni; i++)
   for (j = 0; j < Nj; j++)
    for (k = 0; k < Nk; k++)
// C[i][j] = C[i][j] + A[i][k]*B[k][j];
     C[i*Nj+j]=C[i*Nj+j]+A[i*Nk+k]*B[k*Nj+j];
}

int main(){

  float *h_A, *h_B, *h_C, *h_Cref, *d_A, *d_B, *d_C;
  int i,j,k;
  int Ni, Nj, Nk;

  // int Ni = atoi(argv[1]);
  // int Nj = atoi(argv[2]);
  // int Nk = atoi(argv[3]);

  printf("Specify Matrix dimension Ni, Nj, Nk: ");
  scanf("%d %d %d", &Ni,&Nj,&Nk);

  h_A = (float *) malloc(sizeof(float)*Ni*Nk);
  h_B = (float *) malloc(sizeof(float)*Nk*Nj);
  h_C = (float *) malloc(sizeof(float)*Ni*Nj);
  h_Cref = (float *) malloc(sizeof(float)*Ni*Nj);;

  for (i=0; i<Ni; i++)
   for (k=0; k<Nk; k++)
    h_A[k*Ni+i] = rand();
  for (k=0; k<Nk; k++)
   for (j=0; j<Nj; j++)
    h_B[k*Nj+j] = rand();

  
 // Allocate device memory and copy input data over to GPU
  hipMalloc(&d_A, Ni*Nk*sizeof(float));
  hipMalloc(&d_B, Nk*Nj*sizeof(float));
  hipMalloc(&d_C, Ni*Nj*sizeof(float));
  checkCUDAError("hipMalloc failure");
  hipMemcpy(d_A, h_A, Ni*Nk*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, Nk*Nj*sizeof(float), hipMemcpyHostToDevice);
  checkCUDAError("hipMemcpy H2D transfer failure");

  dim3 block(BLOCK_SIZE,BLOCK_SIZE);  
  dim3 grid(ceil(Ni/(float)BLOCK_SIZE),ceil(Nj/(float)BLOCK_SIZE));
  for(int version=0; version<1; version++)
  {
   for(i=0;i<Ni;i++) for(j=0;j<Nj;j++) h_Cref[i*Nj+j] = 0;
   ab_seq(h_A,h_B,h_Cref,Ni,Nj,Nk);
    for(int trial=0;trial<3;trial++)
    {
     for(i=0;i<Ni;i++) for(j=0;j<Nj;j++) h_C[i*Nj+j] = 0; 
      printf("Trial %d: ",trial);
      hipEventCreate(&start);
      hipEventCreate(&stop);
      hipEventRecord(start);
      // Launch kernel
      switch (version) {
      case 0: ab_gpu<<<grid, block>>>(d_A, d_B, d_C,Ni,Nj,Nk); printf("AB "); break;
      case 1: ab_gpu<<<grid, block>>>(d_A, d_B, d_C,Ni,Nj,Nk); printf("ATB ");break;
      case 2: ab_gpu<<<grid, block>>>(d_A, d_B, d_C,Ni,Nj,Nk); printf("ABT ");break;
      case 3: ab_gpu<<<grid, block>>>(d_A, d_B, d_C,Ni,Nj,Nk); printf("ATBT ");
      }
      checkCUDAError("GPU kernel launch failure");
      hipEventRecord(stop);
      hipEventSynchronize(stop);
      hipEventElapsedTime(&elapsedTime, start,stop);
      hipDeviceSynchronize();
      // Copy results back to host
      hipMemcpy(h_C, d_C, Ni*Nj*sizeof(float), hipMemcpyDeviceToHost);
      checkCUDAError("hipMemcpy D2H");
      for (int i = 0; i < Ni*Nj; i++) if (fabs((h_C[i]-h_Cref[i])/h_Cref[i])>threshold) {printf("Error: mismatch at linearized index %d, was: %f, should be: %f\n", i, h_C[i], h_Cref[i]); return -1;}
      printf("GFLOPS: %.2f\n",2.0e-6*Ni*Nj*Nk/elapsedTime);
     }
  }
  return 0;
}

void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err)
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }
}
