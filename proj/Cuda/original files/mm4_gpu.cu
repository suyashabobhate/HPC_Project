
#include <hip/hip_runtime.h>
__global__ void ab_gpu(const float *__restrict__ A, const float *__restrict__ B, float *__restrict__ C, int Ni, int Nj, int Nk)
{
  int i, j, k;

  for (i = 0; i < Ni; i++) 
   for (j = 0; j < Nj; j++)
    C[i*Nj+j]=0.0;
  for (i = 0; i < Ni; i++)
   for (j = 0; j < Nj; j++)
    for (k = 0; k < Nk; k++)
// C[i][j] = C[i][j] + A[i][k]*B[k][j];
     C[i*Nj+j]=C[i*Nj+j]+A[i*Nk+k]*B[k*Nj+j];
}

__global__ void abT_gpu(const float *__restrict__ A, const float *__restrict__ B, float *__restrict__ C, int Ni, int Nj, int Nk)
{
  int i, j, k;

  for (i = 0; i < Ni; i++) 
   for (j = 0; j < Nj; j++)
    C[i*Nj+j]=0.0;
  for (i = 0; i < Ni; i++)
   for (j = 0; j < Nj; j++)
    for (k = 0; k < Nk; k++)
// C[i][j] = C[i][j] + A[i][k]*B[j][k];
     C[i*Nj+j]=C[i*Nj+j]+A[i*Nk+k]*B[j*Nk+k];
}

__global__ void aTb_gpu(const float *__restrict__ A, const float *__restrict__ B, float *__restrict__ C, int Ni, int Nj, int Nk)
{
  int i, j, k;

  for (i = 0; i < Ni; i++) 
   for (j = 0; j < Nj; j++)
    C[i*Nj+j]=0.0;
  for (i = 0; i < Ni; i++)
   for (j = 0; j < Nj; j++)
    for (k = 0; k < Nk; k++)
// C[i][j] = C[i][j] + A[k][i]*B[k][j];
     C[i*Nj+j]=C[i*Nj+j]+A[k*Ni+i]*B[k*Nj+j];
}

__global__ void aTbT_gpu(const float *__restrict__ A, const float *__restrict__ B, float *__restrict__ C, int Ni, int Nj, int Nk)
{
  int i, j, k;

  for (i = 0; i < Ni; i++) 
   for (j = 0; j < Nj; j++)
    C[i*Nj+j]=0.0;
  for (i = 0; i < Ni; i++)
   for (j = 0; j < Nj; j++)
    for (k = 0; k < Nk; k++)
// C[i][j] = C[i][j] + A[k][i]*B[j][k];
     C[i*Nj+j]=C[i*Nj+j]+A[k*Ni+i]*B[j*Nk+k];
}

